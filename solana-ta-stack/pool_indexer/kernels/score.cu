#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void score_kernel(
    const float* prices,
    const float* volumes,
    float* scores,
    int n,
    const float* weights
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        if (idx > 0 && prices[idx-1] > 1e-9 && volumes[idx-1] > 1e-9) {
            // Factor 1: Price change (momentum)
            float price_change = (prices[idx] - prices[idx-1]) / prices[idx-1];
            
            // Factor 2: Volume change
            float volume_change = (volumes[idx] - volumes[idx-1]) / volumes[idx-1];

            // Final weighted score
            scores[idx] = (weights[0] * price_change) + (weights[1] * volume_change);
        } else {
            scores[idx] = 0.0f;
        }
    }
}

// Additional kernels for technical indicators
extern "C" __global__ void sma_kernel(
    const float* values,
    float* sma,
    uint32_t n,
    uint32_t window
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n || idx < window - 1) return;
    
    float sum = 0.0f;
    for (uint32_t i = 0; i < window; i++) {
        sum += values[idx - i];
    }
    sma[idx] = sum / window;
}
